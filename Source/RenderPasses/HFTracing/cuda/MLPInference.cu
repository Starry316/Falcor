#include "MLPInference.h"

// A wrapper function that launches the kernel.
void launchCopySurface(hipSurfaceObject_t input, hipSurfaceObject_t output, unsigned int width, unsigned int height, unsigned int format)
{
    dim3 dimBlock(16, 16);
    dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);

}
